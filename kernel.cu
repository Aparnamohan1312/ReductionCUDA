
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512


__global__ void naiveReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    // NAIVE REDUCTION IMPLEMENTATION
  __shared__ float bl[2*BLOCK_SIZE];
    unsigned int id = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2;
    
    if ( id+i >= size)
        bl[i] = 0.0;
    else
        bl[id] = in[id+i];

    if (i + id + blockDim.x >= size)
        bl[id + blockDim.x] = 0.0;
    else
        bl[id+ blockDim.x] = in[id + i+ blockDim.x];
    

    for (unsigned int j=1; j<= blockDim.x; j= j*2) 
    {
       __syncthreads();
       
       if (id % j == 0) 
           bl[id*2] += bl[id*2 + j];
    }

    if (id == 0) 
          out[blockIdx.x] = bl[0];

}

__global__ void optimizedReduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    // OPTIMIZED REDUCTION IMPLEMENTATION
 __shared__ float bl[BLOCK_SIZE*2];
    unsigned int id = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2;

    if (id + i >= size)
        bl[id] = 0.0;
    else
        bl[id] = in[id+i];

    if (i + id + blockDim.x >= size)
        bl[id+ blockDim.x] = 0.0;
    else
        bl[ blockDim.x+id] = in[blockDim.x+id+i];


    for (unsigned int j=blockDim.x; j> 0 ; j=j/2)
    {
       __syncthreads();

       if (id< j)
           bl[id] = bl[id] +  bl[j+id];
    }

    if (id== 0) 
	out[blockIdx.x] = bl[0];


}
